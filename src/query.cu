#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++) {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
        printf("Device %d compute capability: %d.%d\n", i, devProp.major, devProp.minor);
    }
    return 0;
}

